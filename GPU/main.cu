#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
//																		//
//		GPU accelerated max flow min cut graph problem solver			//
//																		//
//		Written by: Apoorva Gupta										//
//					Jorge Salazar										//
//					Jiho Yang											//
//																		//
//		Final update: 09/10/2017										//
//																		//
//////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <vector>
#include <time.h>
#include "read_bk.h"
#include "primal_dual.cuh"
#include "mathOperations.cuh"
#include "postProcessing.cuh"
#include "helper.cuh"
#include <string.h>	
#include <hipblas.h>
#include <cassert>
#include <unistd.h>

//# define T float
//# define FLOAT
#define T double
#define DOUBLE

using namespace std;

template<class S>
void cudaAssert(S* d_arr, int num_elem, char*s)
{
	S* temp = new S[num_elem];
	hipMemcpy(temp, d_arr, num_elem*sizeof(S), hipMemcpyDeviceToHost);
	cout << "Assertion for "<<s<<endl;
	for(int i = 0; i<num_elem; i++)
	{
		assert(!std::isnan(temp[i]));
	} 
	cout << "Assertion passed"<<endl;
	delete temp; 	
}

template<class S>
void printDevice(S* d_arr, int num_elem, char* s)
{
	S* temp = new S[num_elem] ;
	hipMemcpy(temp, d_arr, num_elem*sizeof(S), hipMemcpyDeviceToHost);
	for(int i = 0; i<num_elem; i++)
	{
		cout<< s << "_"<<i<<" is "<< temp[i] <<endl; 
	} 
	delete temp; 
}

int main(int argc, char **argv)
{
    if (argc <= 1)
	{
		printf("Usage: %s <filename> -alpha <value> - rho <value> -it <maximum number of iterations>\n", argv[0]);
		return 1;
    }
	// Start time
	clock_t tStart_total = clock();
	clock_t tStart_load = clock();

	// Parameters
	T alpha = 1;
	T rho = 1;
	T gap = 1;
	T eps = 1E-10;
	int it  = 0;
	int iter_max = 100000;
	T xf;
	T x_norm;
	T max_flow;
	T max_val;

	// Command line parameters
	cout << argv[1] << endl;
	getParam("alpha", alpha, argc, argv);
	cout << "alpha: " << alpha << endl;
	getParam("rho", rho, argc, argv);
	cout << "rho: " << rho << endl;
	getParam("it", iter_max, argc, argv);
	cout << "it: " << iter_max << endl;

	// Import bk file    
	read_bk<T> *g = new read_bk<T>(argv[1]); 	
	int numNodes  = g->nNodes;
	int numEdges = g->nEdges;
	T *f = g->f;
	T *w = g->w;
	T b = g->b;
	int* start_edge = g->edge_start;
	int* end_edge = g->edge_end;
	int* d_start_edge , *d_end_edge;
	int double_edges = 2 * numEdges;
	int* h_nbhd_size = &(g->h_nbhd_size[0]); 
	int* h_nbhd_start = &(g->h_nbhd_start[0]) ;
 	int* h_nbhd_vert = &(g->h_nbhd_vert[0]);
 	int *h_nbhd_sign = &(g->h_nbhd_sign[0]);
	int *h_nbhd_edges = &(g->h_nbhd_edges[0]);
 	int *d_nbhd_size, *d_nbhd_start, *d_nbhd_vert, *d_nbhd_sign, *d_nbhd_edges;
	T *d_f , *d_w;

	// End of CPU loading time
	clock_t tEnd_load = clock();

	// Names of all the cuda_arrays	
 	T *d_x, *d_y, *d_div_y, *d_x_diff, *d_grad_x_diff, *d_tau, *d_sigma;
 	T *d_grad_x, *d_max_vec, *d_gap_vec;
	
	// Start time for CUDA copy
	clock_t tStart_copy = clock();
	
	// Allocate memory on CUDA
	hipMalloc((void**)&d_f , numNodes*sizeof(T));																//CUDA_CHECK;
	hipMalloc((void**)&d_w , numEdges*sizeof(T));																//CUDA_CHECK;
	hipMalloc((void**)&d_start_edge , numEdges*sizeof(int));													//CUDA_CHECK;
	hipMalloc((void**)&d_end_edge , numEdges*sizeof(int));														//CUDA_CHECK;
 	hipMalloc((void**)&d_nbhd_size , numNodes*sizeof(int));													//CUDA_CHECK;
 	hipMalloc((void**)&d_nbhd_start , numNodes*sizeof(int));													//CUDA_CHECK;
	hipMalloc((void**)&d_nbhd_vert , double_edges*sizeof(int));												//CUDA_CHECK;
	hipMalloc((void**)&d_nbhd_sign , double_edges*sizeof(int));												//CUDA_CHECK;
	hipMalloc((void**)&d_nbhd_edges , double_edges*sizeof(int)); 												//CUDA_CHECK;
	hipMalloc((void**)&d_x, numNodes*sizeof(T));																//CUDA_CHECK;
	hipMalloc((void**)&d_y, numEdges*sizeof(T));																//CUDA_CHECK;
	hipMalloc((void**)&d_div_y, numNodes*sizeof(T));															//CUDA_CHECK;
	hipMalloc((void**)&d_x_diff, numNodes*sizeof(T));															//CUDA_CHECK;
	hipMalloc((void**)&d_grad_x_diff, numEdges*sizeof(T));														//CUDA_CHECK;
	hipMalloc((void**)&d_tau, numNodes*sizeof(T));																//CUDA_CHECK;
	hipMalloc((void**)&d_sigma, numEdges*sizeof(T));															//CUDA_CHECK;
	hipMalloc((void**)&d_grad_x, numEdges*sizeof(T));															//CUDA_CHECK;
	hipMalloc((void**)&d_max_vec, numNodes*sizeof(T));															//CUDA_CHECK;
	hipMalloc((void**)&d_gap_vec, numNodes*sizeof(T));															//CUDA_CHECK;
	
	// Copy memory to device
	hipMemcpy(d_f , f, numNodes*sizeof(T), hipMemcpyHostToDevice);											//CUDA_CHECK;
	hipMemcpy(d_w , w, numEdges*sizeof(T), hipMemcpyHostToDevice);											//CUDA_CHECK;
	hipMemcpy(d_start_edge , start_edge, numEdges*sizeof(int), hipMemcpyHostToDevice);						//CUDA_CHECK;
	hipMemcpy(d_end_edge , end_edge, numEdges*sizeof(int), hipMemcpyHostToDevice);							//CUDA_CHECK;
	hipMemcpy(d_nbhd_size , h_nbhd_size, numNodes*sizeof(int), hipMemcpyHostToDevice);						//CUDA_CHECK;
	hipMemcpy(d_nbhd_start , h_nbhd_start, numNodes*sizeof(int), hipMemcpyHostToDevice);						//CUDA_CHECK;
	hipMemcpy(d_nbhd_vert , h_nbhd_vert, double_edges*sizeof(int), hipMemcpyHostToDevice);					//CUDA_CHECK;
	hipMemcpy(d_nbhd_sign , h_nbhd_sign, double_edges*sizeof(int), hipMemcpyHostToDevice);					//CUDA_CHECK;
	hipMemcpy(d_nbhd_edges , h_nbhd_edges, double_edges*sizeof(int), hipMemcpyHostToDevice);					//CUDA_CHECK;
	
	// Initialise cuda memories
	hipMemset(d_x , 0, numNodes*sizeof(T));																	//CUDA_CHECK;
	hipMemset(d_y , 0, numEdges*sizeof(T));																	//CUDA_CHECK;
	hipMemset(d_div_y , 0, numNodes*sizeof(T));																//CUDA_CHECK;
	hipMemset(d_x_diff , 0, numNodes*sizeof(T));																//CUDA_CHECK;
	hipMemset(d_grad_x_diff , 0, numEdges*sizeof(T));															//CUDA_CHECK;
	hipMemset(d_tau , 1, numNodes*sizeof(T));																	//CUDA_CHECK;
	hipMemset(d_sigma , 1, numEdges*sizeof(T));																//CUDA_CHECK;
	hipMemset(d_grad_x, 0 , numEdges*sizeof(T));																//CUDA_CHECK;
	hipMemset(d_max_vec, 0 , numNodes*sizeof(T));																//CUDA_CHECK;
	hipMemset(d_gap_vec, 0 , numNodes*sizeof(T));																//CUDA_CHECK;
	
	// End time for CUDA copy
	clock_t tEnd_copy = clock();
	
	// cuBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	// GPU Block configuration
	dim3 block = dim3(1024,1,1);
	int grid_x = ((max(numNodes, numEdges) + block.x - 1)/block.x);
	int grid_y = 1;
	int grid_z = 1;
	dim3 grid = dim3(grid_x, grid_y, grid_z);
	
	// Start time for GPU computation
	clock_t tStart_comp = clock();
	
	// Compute dt
	d_compute_dt <<<grid, block>>> (d_tau, d_sigma, d_w, alpha, rho, d_nbhd_size, d_nbhd_edges, d_nbhd_start, numNodes, numEdges); 									//CUDA_CHECK;
	
	// Main Iteration
	while (it < iter_max && gap > eps){
	
		// Update X
		updateX <T> <<< grid, block >>> (d_x, d_y, d_w, d_f, d_x_diff, d_div_y, d_nbhd_size, d_nbhd_start, d_nbhd_sign, d_nbhd_edges, d_tau, numNodes);				//CUDA_CHECK;
	
		// Update Y
		updateY <T> <<<grid, block >>> (d_x_diff, d_y, d_w, d_start_edge, d_end_edge, d_sigma, numEdges);															//CUDA_CHECK;
	
		// Compute gap once in every 100 iterations
		if (it % 100 == 0){
	
			// Update divergence of Y	
			h_divergence_calculate <T> <<<grid, block>>> (d_w, d_y, d_nbhd_size, d_nbhd_start, d_nbhd_sign, d_nbhd_edges, numNodes, d_div_y);						//CUDA_CHECK;
	
			// Compare 0 and div_y - f
			max_vec_computation <T> <<<grid, block >>> (d_div_y, d_f, d_max_vec, numNodes);  																		//CUDA_CHECK;
	
			// Compute gradient of u
			h_gradient_calculate <T> <<<grid, block>>>(d_w, d_x, d_start_edge, d_end_edge, numEdges, d_grad_x);														//CUDA_CHECK;
			#ifdef FLOAT
	
				// Compute L1 norm of gradient of u
				hipblasSasum(handle, numEdges, d_grad_x, 1, &x_norm);  											//CUDA_CHECK;
	
				// Compute scalar product 
				hipblasSdot(handle, numNodes, d_x, 1, d_f, 1, &xf);	                                			//CUDA_CHECK;
	
				// Summing up the max_vec
				hipblasSasum(handle, numNodes, d_max_vec, 1, &max_val); 											//CUDA_CHECK;
			#else
				// Compute L1 norm of gradient of u
				hipblasDasum(handle, numEdges, d_grad_x, 1, &x_norm);											//CUDA_CHECK;
				
				// Compute scalar product 
				hipblasDdot(handle, numNodes, d_x, 1, d_f, 1, &xf);												//CUDA_CHECK;
				
				// Compute scalar product 
				hipblasDasum(handle, numNodes, d_max_vec, 1, &max_val);											//CUDA_CHECK;
			#endif
			
			// Compute gap
			gap = (xf + x_norm + max_val) / (T)numEdges;
		}
		it = it + 1;
	}
	
	// End computation time
	clock_t tEnd_comp = clock();
	// Round solution
	round_solution <T> <<<grid, block>>> (d_x, numNodes);														//CUDA_CHECK;
	// Compute max flow
	h_gradient_calculate <T> <<<grid, block>>>(d_w, d_x, d_start_edge, d_end_edge, numEdges, d_grad_x);			//CUDA_CHECK;

	#ifdef FLOAT
		hipblasSasum(handle, numEdges, d_grad_x, 1, &x_norm);  													//CUDA_CHECK;
		hipblasSdot(handle, numNodes, d_x, 1, d_f, 1, &xf);	                           							//CUDA_CHECK;
	#else
		hipblasDasum(handle, numEdges, d_grad_x, 1, &x_norm);													//CUDA_CHECK;
		hipblasDdot(handle, numNodes, d_x, 1, d_f, 1, &xf);														//CUDA_CHECK;
	#endif
	// Compute max flow
	max_flow = xf + x_norm + b;
	// Print max flow
	cout << fixed << "Max flow = " << (int)max_flow << endl;
	// End of total Execution
	clock_t tEnd_total = clock();
	// Program exit messages
	if (it == iter_max) cout << "ERROR: Maximum number of iterations reached" << endl << endl;
	cout << "cpu_load_time: " << (double)1000*(tEnd_load - tStart_load)/CLOCKS_PER_SEC << " ms" << endl;
	cout << "gpu_load_time: " << (double)1000*(tEnd_copy - tStart_copy)/CLOCKS_PER_SEC << " ms" << endl;
	cout << "gpu_compute_time: " << (double)1000*(tEnd_comp - tStart_comp)/CLOCKS_PER_SEC << " ms" << endl;
	cout << "Total execution time: " << (double)1000*(tEnd_total - tStart_total)/CLOCKS_PER_SEC << " ms" << endl;
	
	// Output the start_edge and end_edge arrays

	char buffer[500];
	char *answer = getcwd(buffer, sizeof(buffer));
	string s_cwd;
	if (answer)
	{
	    s_cwd = answer;
	}

	string file_name = s_cwd + "/" + "start_edge.txt";
	output_data(argv, start_edge, numEdges, file_name);
	file_name = s_cwd + "/" + "end_edge.txt";
	output_data(argv, end_edge, numEdges, file_name);

	// Free memory    
	delete g;
	hipFree(d_f);																								//CUDA_CHECK;
	hipFree(d_w);																								//CUDA_CHECK;
	hipFree(d_start_edge);																						//CUDA_CHECK;
	hipFree(d_end_edge);																						//CUDA_CHECK;
	hipFree(d_nbhd_size);																						//CUDA_CHECK;
	hipFree(d_nbhd_start);																						//CUDA_CHECK;
	hipFree(d_nbhd_vert);																						//CUDA_CHECK;
	hipFree(d_nbhd_sign);																						//CUDA_CHECK;
	hipFree(d_nbhd_edges);																						//CUDA_CHECK;
	hipFree(d_x);																								//CUDA_CHECK;
	hipFree(d_y);																								//CUDA_CHECK;
	hipFree(d_div_y);																							//CUDA_CHECK;
	hipFree(d_x_diff);																							//CUDA_CHECK;
	hipFree(d_grad_x_diff);																					//CUDA_CHECK;
	hipFree(d_tau);																							//CUDA_CHECK;
	hipFree(d_sigma);																							//CUDA_CHECK;
	hipFree(d_grad_x);																							//CUDA_CHECK;
	hipFree(d_max_vec);																						//CUDA_CHECK;
	hipFree(d_gap_vec);																						//CUDA_CHECK;

    return 0;
}
